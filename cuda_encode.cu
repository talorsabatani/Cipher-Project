#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cipher.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#define NUM_OF_THREADS 1024  // maximum number of threads per block

// run on the kernel
__global__ void addCalculateKernel(char *cipher_text, int text_len, unsigned char *key, int key_len_as_byts, int* d_global_all_ascii) {
	int tid = threadIdx.x;
	int charId = tid;

	// shared memory is allocated between threads in a block
	__shared__ int shared_non_ascii_counter;
	__shared__ int maximum_non_ascii_allowed;
	//__shared__ unsigned int key_as_int;

	if (tid == 0) {
		shared_non_ascii_counter = 0; 
		maximum_non_ascii_allowed = text_len * MAX_PERCENTAGE_NON_PRINTABLE; // we are allowing some percentage of not-printable ascii
		if (maximum_non_ascii_allowed <= 0) {
			maximum_non_ascii_allowed = 1;
		}
		/*
		key_as_int = 0;
		for (int i = 0; i < key_len_as_byts; i++) {
			key_as_int *= 256;
			key_as_int += key[i];
		} */
	}

	// sync threads in the same block
	__syncthreads();

	while (charId < text_len) { // out of the borders of the text
		if (shared_non_ascii_counter < maximum_non_ascii_allowed) { // checks if the number of non-ascii allowed not reached to the maximum
			cipher_text[charId] ^=  key[charId % (key_len_as_byts)]; 

			//if isn't printable ascii characters		
			// 13 and 10 together are "\n". 32 to 126 are all the printable ascii characters.
			if (!(cipher_text[charId] == LINE_FEED || cipher_text[charId] == CARRIAGE_RETURN || (MIN_PRINTABLE <= cipher_text[charId] && cipher_text[charId] <= MAX_PRINTABLE))) {
				//if (key_as_int != 111) 	
				atomicAdd(&shared_non_ascii_counter, 1); // used only one block at a time
				//else
				//	printf("key %u, char id: %d, not good int: %u\n", key_as_int, charId, (unsigned char)cipher_text[charId]);
			}

		}

		charId += NUM_OF_THREADS;
	}

	// sync threads
	__syncthreads();
	
	if (tid == 0) {
		// checks if the number of non-ascii allowed reached to the maximum
		if (shared_non_ascii_counter > maximum_non_ascii_allowed) {
			*d_global_all_ascii = 0;
		}
	}

}

int cuda_calc_plain(char *cipher_text, int text_len, unsigned int key_as_int, int key_len_as_byts, char** plain_text) {

	char *d_temp_text = 0;
	unsigned char* d_temp_key = 0;
	int* d_global_all_ascii = 0;
	int cpu_all_ascii = 1;
	int num_block = 1;// text_len / NUM_OF_THREADS + 1;
	hipError_t cudaStatus;
	int i = key_len_as_byts-1; 

	*plain_text = (char*)calloc(text_len, sizeof(char));
	if (!(*plain_text)) {
		printf("calloc for plain_text faliled.");
		return -1;
	}

	unsigned char* key = (unsigned char*)calloc(key_len_as_byts, sizeof(unsigned char));
	if (!key) {
		printf("calloc for key faliled.");
		return -1;
	}

	while (key_as_int > 0) {
		key[i] = (unsigned char) (key_as_int % 256); // get the first right byte into the key[i]
		key_as_int /= 256; // remove the first right byte
		i--;
	}		

	// Choose which GPU to run on
	cudaStatus = hipSetDevice(0);

	// allocated space - create d_global_all_ascii for cuda
	cudaStatus = hipMalloc((void**) &d_global_all_ascii, sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc all_ascii failed.");
		return -1;
	}

	//reset to 1 in all_ascii in gpu
	cudaStatus = hipMemcpy(d_global_all_ascii, &cpu_all_ascii, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed.");
		return -1;
	}


	// create d_temp_text for cuda
	cudaStatus = hipMalloc((void**) &d_temp_text, text_len * sizeof(char));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc d_temp_text failed.");
		return -1;
	}

	//Copy cipher text from cpu to gpu
	cudaStatus = hipMemcpy(d_temp_text, cipher_text, text_len * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed.");
		return -1;
	}

	// create d_temp_key for cuda
	cudaStatus = hipMalloc((void**) &d_temp_key, key_len_as_byts * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc d_temp_key failed.");
		return -1;
	}

	//Copy key from cpu to gpu
	cudaStatus = hipMemcpy(d_temp_key, key, key_len_as_byts * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed.");
		return -1;
	}

	
	//Launch a kernel on the GPU with 1024 threads for every block
	// num_block = text_len / NUM_OF_THREADS +1
	// NUM_OF_THREADS = 1024
	addCalculateKernel<<<num_block, NUM_OF_THREADS>>>(d_temp_text, text_len, d_temp_key, key_len_as_byts, d_global_all_ascii);

	//Copy all_ascii from gpu to cpu
	cudaStatus = hipMemcpy(&cpu_all_ascii, d_global_all_ascii, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed.");
		return -1;
	}

	if(cpu_all_ascii){
		//Copy plain text from gpu to cpu
		cudaStatus = hipMemcpy(*plain_text, d_temp_text, text_len * sizeof(char), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			printf("hipMemcpy failed.");
			return -1;
		}
	}

	// free memory
	hipFree(d_temp_text);
	hipFree(d_temp_key);
	hipFree(d_global_all_ascii);
	free(key);

	
	// return the status that says if all is printable ascii (0-no 1-yes)
	return cpu_all_ascii;
}

